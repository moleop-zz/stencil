
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 32

typedef struct {
  int width;
  int height;
  int stride;
  float* elem;
} Matrix;

__device__ Matrix GetSubMatrix(Matrix A, int row, int col){

  Matrix submat;
  submat.width = BLOCKSIZE;
  submat.height = BLOCKSIZE;
  submat.stride = A.stride;
  submat.elem = &A.elem[A.stride * BLOCKSIZE * row + BLOCKSIZE * col];
  return submat;
}

__device__ void SetElement(Matrix A, int row, int col, float value){
  A.elem[row * A.stride + col] = value;
 }

__device__ float GetElement(const Matrix A, int row, int col){
  return A.elem[row * A.stride + col];
}

__device__ float GetNextValue(float Mat[BLOCKSIZE][BLOCKSIZE], int row, int col, float newValue){
  if (row != 0 && row != BLOCKSIZE-1 && col != 0 && col!=BLOCKSIZE-1){
    newValue = Mat[row][col] + Mat[row-1][col] + Mat[row+1][col] +Mat[row][col-1] + Mat[row][col+1];
  }else{
    newValue = Mat[row][col];
    if (row == 0){  newValue += Mat[row+1][col];
      if (col == 0) newValue += Mat[row][col+1];
      else if (col == BLOCKSIZE-1) newValue += Mat[row][col-1];
      else newValue += Mat[row][col+1] + Mat[row][col-1];
    }
    else if (row == BLOCKSIZE-1) {newValue += Mat[row-1][col];
      if (col == 0) newValue += Mat[row][col+1];
      else if (col == BLOCKSIZE-1) newValue += Mat[row][col-1];
      else newValue += Mat[row][col+1] + Mat[row][col-1];
    }
    else if (col == 0) newValue += Mat[row][col+1] + Mat[row+1][col] + Mat[row-1][col];
    else newValue += Mat[row][col-1] + Mat[row+1][col] + Mat[row-1][col];
  }
  return newValue/5;
  }


__global__ void StencilKernel(Matrix A){

  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  float newValue = 0;

  int row = threadIdx.y;
  int col = threadIdx.x;

  Matrix submat;

  for (int k = 0; k < (A.width/ BLOCKSIZE); ++k){
    submat = GetSubMatrix(A, blockRow, k);
    __shared__ float Mat[BLOCKSIZE][BLOCKSIZE];
    Mat[row][col] = GetElement(submat, row, col);
    __syncthreads();

    newValue = GetNextValue(Mat, row, col, newValue);
    __syncthreads();
  }
  SetElement(submat, row, col, newValue);
}

__host__ void init_mat(Matrix A){
  for (int i = 0; i < A.width*A.height; ++i){
    A.elem[i] = 1;
  }
}

__host__ void print_mat(Matrix A){
  for (int i = 0; i < A.width*A.height; ++i){
    if (i % A.width == 0) printf("\n");
    printf("%.3f ", A.elem[i]);
  }
}

int main(int argc, char const *argv[]) {

  Matrix h_mat;
  h_mat.width = 32;
  h_mat.height = 32;
  h_mat.stride = h_mat.width;
  //int N = h_mat.width * h_mat.height;
  int size = h_mat.width * h_mat.height * sizeof(float);
  h_mat.elem =(float*)malloc(size);

  Matrix d_mat;
  d_mat.width = h_mat.width;
  d_mat.height = h_mat.height;
  d_mat.stride = d_mat.width;
  hipMalloc(&d_mat.elem, size);

  init_mat(h_mat);
  hipMemcpy(d_mat.elem, h_mat.elem, size, hipMemcpyHostToDevice);

  dim3 threads(BLOCKSIZE, BLOCKSIZE); // 2 dimensional  
  dim3 grid (h_mat.width / threads.x, h_mat.height / threads.y);
  for (int run = 0; run < 5;++run){
  StencilKernel<<<grid,threads>>>(d_mat);
  hipMemcpy(h_mat.elem, d_mat.elem, size, hipMemcpyDeviceToHost);
  }
  print_mat(h_mat);

  hipFree (d_mat.elem);
  free (h_mat.elem);


  return 0;
}
