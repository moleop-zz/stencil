
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 16

typedef struct {
  int width;
  int height;
  int stride;
  float* elem;
} Matrix;

__device__ Matrix GetSubMatrix(Matrix A, int row, int col, int num_gh){

  Matrix submat;
  submat.width = BLOCKSIZE;
  submat.height = BLOCKSIZE;
  submat.stride = A.stride;
  submat.elem = &A.elem[A.stride * (BLOCKSIZE-2-num_gh) * row   + (BLOCKSIZE-2-num_gh) * col];

  return submat;
}

__device__ void SetElement(Matrix A, int row, int col, float value){
  A.elem[row * A.stride + col] = value;
 }

__device__ float GetElement(const Matrix A, int row, int col, int blockRow, int blockCol, dim3 grid, int size){
  if (blockRow < grid.y-1 && blockCol < grid.x-1){
    return A.elem[row * A.stride + col];
  // Ausnahme für submatrix die am Rand liegt ... Auffüllen mit 0.
  }else{
    float tmp[BLOCKSIZE*BLOCKSIZE];
    if (blockCol*(BLOCKSIZE-2)+col < size && blockRow*(BLOCKSIZE-2)+row < size){
      tmp[row*(BLOCKSIZE-1)+col] = A.elem[row * A.stride + col];
    }else tmp[row*(BLOCKSIZE-1)+col] = 0;
    return tmp[row*(BLOCKSIZE-1)+col];

  }
}

__device__ float GetNextValue(float Mat[BLOCKSIZE][BLOCKSIZE], int row, int col, float newValue){
  if (row != 0 && row != BLOCKSIZE-1 && col != 0 && col!=BLOCKSIZE-1){
    newValue = Mat[row][col] + Mat[row-1][col] + Mat[row+1][col] +Mat[row][col-1] + Mat[row][col+1];
  }else{
    newValue = Mat[row][col];
    if (row == 0){  newValue += Mat[row+1][col];
      if (col == 0) newValue += Mat[row][col+1];
      else if (col == BLOCKSIZE-1) newValue += Mat[row][col-1];
      else newValue += Mat[row][col+1] + Mat[row][col-1];
    }
    else if (row == BLOCKSIZE-1) {newValue += Mat[row-1][col];
      if (col == 0) newValue += Mat[row][col+1];
      else if (col == BLOCKSIZE-1) newValue += Mat[row][col-1];
      else newValue += Mat[row][col+1] + Mat[row][col-1];
    }
    else if (col == 0) newValue += Mat[row][col+1] + Mat[row+1][col] + Mat[row-1][col];
    else newValue += Mat[row][col-1] + Mat[row+1][col] + Mat[row-1][col];
  }
  return newValue/5;
  }


__global__ void StencilKernel(Matrix A, dim3 grid, int num_gh, int size){

  int blockRow = blockIdx.y;
 // printf("%d", blockRow);
  int blockCol = blockIdx.x;

  float newValue = 0;

  int row = threadIdx.y;
  int col = threadIdx.x;

  //printf("%d ", row);
  Matrix submat;

    submat = GetSubMatrix(A, blockRow, blockCol, num_gh);
    __shared__ float Mat[BLOCKSIZE][BLOCKSIZE];


    Mat[row][col] = GetElement(submat, row, col, blockRow, blockCol, grid, size);
    __syncthreads();

    newValue = GetNextValue(Mat, row, col, newValue);
    __syncthreads();

  if (blockRow == 0 && blockCol == 0 ){
  	if (row != BLOCKSIZE-1   && col != BLOCKSIZE-1){
  		SetElement(submat, row, col, newValue);
  	}
  }else if (blockRow == 0 && blockCol != 0 && blockCol < grid.x-1){
   	if (col > 0 && col < BLOCKSIZE-1 && row != BLOCKSIZE-1) // letzter Teil kann wahrscheinlich weg
   		SetElement(submat, row, col, newValue);

  }else if (blockCol == 0 && blockRow != 0 && blockRow < grid.y-1){
   	if (row > 0 && row < BLOCKSIZE-1 && col != BLOCKSIZE-1 )
   		SetElement(submat, row, col, newValue);

	// hier weiter mache

  }else if (blockCol == grid.x-1 && blockRow == 0){
    if (row < BLOCKSIZE -1 && col > 0 && blockCol*(BLOCKSIZE-2)+col < size )
      SetElement(submat, row, col, newValue);

  }else if (blockCol == grid.x-1 && blockRow != 0){
    if (row > 0 && row < BLOCKSIZE -1 && col > 0 && blockCol*(BLOCKSIZE-2)+col < size )
      SetElement(submat, row, col, newValue);

// unten und unten links

  }else if (blockRow == grid.y-1 && blockCol == 0){
      if (col < BLOCKSIZE -1 && row > 0 && blockRow*(BLOCKSIZE-2)+row < size )
        SetElement(submat, row, col, newValue);

  }else if (blockRow == grid.y-1 && blockCol != 0){
      if (col> 0 && col < BLOCKSIZE -1 && row > 0 && blockRow*(BLOCKSIZE-2)+row < size )
        SetElement(submat, row, col, newValue);

  }else if (blockCol != 0 && blockRow != 0 && blockCol != grid.x-1 && blockRow != grid.y-1){
     	if (col != BLOCKSIZE-1 && row != 0 && row != BLOCKSIZE-1 && col != 0 )
   		SetElement(submat, row, col, newValue);
  }
}

__host__ void init_mat(Matrix A){
  for (int i = 0; i < A.width*A.height; ++i){
    A.elem[i] = 1;
  }
}

__host__ void print_mat(Matrix A){
  for (int i = 0; i < A.width*A.height; ++i){
    if (i % A.width == 0) printf("\n");
    printf("%.3f ", A.elem[i]);
  }
}

int main(int argc, char const *argv[]) {

  //Größe des Feldes
  int size=32;
  //Anzahl Iterationen
  int iter=3;
  //Anzahl der Ghostcells (Überlapp)
  int num_gh = 0;
  // interation ohne exchange
  //int iter2=std::ceil(iter/num_gh);
  //Ausgabedatei
  char *filename="out.ppm";

  Matrix h_mat;
  h_mat.width = size;
  h_mat.height = size;
  h_mat.stride = h_mat.width;
  int mem = h_mat.width * h_mat.height * sizeof(float);
  h_mat.elem =(float*)malloc(mem);

  Matrix d_mat;
  d_mat.width = h_mat.width;
  d_mat.height = h_mat.height;
  d_mat.stride = d_mat.width;
  hipMalloc(&d_mat.elem, mem);

  init_mat(h_mat);
  hipMemcpy(d_mat.elem, h_mat.elem, mem, hipMemcpyHostToDevice);

  dim3 threads(BLOCKSIZE, BLOCKSIZE); // 2 dimensional
  dim3 grid (std::ceil((double)h_mat.width / (threads.x-2)),std::ceil((double)h_mat.height / (threads.y-2)));
  printf("grid.x = %d\n", grid.x);
  printf("gridy = %d\n", grid.y);
  for (int run = 0; run < iter;++run){
    StencilKernel<<<grid,threads>>>(d_mat, grid, num_gh, size);
  }
  hipMemcpy(h_mat.elem, d_mat.elem, mem, hipMemcpyDeviceToHost);
  print_mat(h_mat);

  hipFree (d_mat.elem);
  free (h_mat.elem);


  return 0;
}
